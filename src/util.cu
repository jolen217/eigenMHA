#include "hip/hip_runtime.h"

#include "util.cuh"

std::function<bool(float,float,float)> NEAR2 = [](float a, float b, float prec) -> bool { return ((a != a && b != b) 
  || (a == std::numeric_limits<typename std::remove_reference<decltype(a)>::type>::infinity() 
    && b == std::numeric_limits<typename std::remove_reference<  decltype(b)>::type>::infinity()) 
  || (-a == std::numeric_limits<typename std::remove_reference< decltype(a)>::type>::infinity() 
    && -b == std::numeric_limits<typename std::remove_reference<  decltype(b)>::type>::infinity()) 
  || (abs(a - b) / abs(a) < prec) || (abs(a - b) / abs(b) < prec) || (abs(a - b) < prec)); };

void print_vec(const float *outv, std::string outn, int start, int end) {
std::cout << outn << ": ";
for(int i=start; i<end; i++) {
  std::cout << outv[i] << " ";
}
std::cout << std::endl;
}

bool compareResults(const float *res, const float *ref, int len) {
  bool is_near2 = true;
  for (unsigned int i = 0; i < len; i++) {
    // if(!NEAR2(static_cast<float>(res[i]), ref[i], 1e-1)){
    //   std::cout << i << ": " << res[i] << " " << ref[i] << std::endl;
    // }
      is_near2 &= NEAR2(static_cast<float>(res[i]), ref[i], 1e-1);
  }
  return is_near2;
}

void checkCudaError(hipError_t code, const char *expr, const char *file, int line) {
  if (code) {
      fprintf(stderr, "ERROR: CUDA error at %s:%d, code=%d (%s) in '%s'\n\n",
              file, line, (int)code, hipGetErrorString(code), expr);
      exit(1);
  }
}

void checkCudnnError(hipdnnStatus_t code, const char *expr, const char *file, int line) {
  if (code) {
      fprintf(stderr, "CUDNN error at %s:%d, code=%d (%s) in '%s'\n\n",
              file, line, (int)code, hipdnnGetErrorString(code), expr);
      exit(1);
  }
}

/* mse loss kernel
@ target, output, d_loss are is 2d data [batch,out_features]
@ loss is a scalar
*/
__global__ void mse_loss_kernel(const float* output, const float* target, float* loss, float* d_loss, int num_elem){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if(idx==0) *loss=0;

  if(idx<num_elem)
  {
      float err = output[idx] - target[idx];
      float err2 = err * err;
      float mean_square_error = err2/num_elem;
      atomicAdd(loss, mean_square_error); // poor performance
      d_loss[idx] = 2 * err * (1.0f/num_elem);
  }
}

void launch_mse_loss_kernel(const float* output, const float* target, float* loss, float* d_loss, int num_elem){
  dim3 blocks((num_elem - 1) / 512 + 1);
  dim3 threads(512);
  mse_loss_kernel<<<blocks, threads>>>(output, target, loss, d_loss, num_elem);
  CHECK_CUDA_ERR(hipDeviceSynchronize());
}

std::vector<float> vector0213(std::vector<float> data, int A, int B, int C, int D){
  assert(data.size()==A*B*C*D);
  std::vector<float> temp_data = data;
  for(int a=0;a<A;a++)
  for(int b=0;b<B;b++)
  for(int c=0;c<C;c++)
  for(int d=0;d<D;d++){
    temp_data.at(a*(B*C*D)+(c*B+b)*D+d) = data.at(a*(B*C*D)+(b*C+c)*D+d);
  }
  return temp_data;
}

std::vector<float> vector0132(std::vector<float> data, int A, int B, int C, int D){
  assert(data.size()==A*B*C*D);
  std::vector<float> temp_data = data;
  for(int a=0;a<A;a++)
  for(int b=0;b<B;b++)
  for(int c=0;c<C;c++)
  for(int d=0;d<D;d++){
    temp_data.at(a*(B*C*D)+(b*D+d)*C+c) = data.at(a*(B*C*D)+(b*C+c)*D+d);
  }
  return temp_data;
}

std::vector<float> vector01(std::vector<float> data, int A, int B){
  assert(data.size()==A*B);
  std::vector<float> temp_data = data;
  for(int a=0;a<A;a++)
    for(int b=0;b<B;b++){
      temp_data.at(b*A+a) = data.at(a*B+b);
    }
  return temp_data;
}


std::vector<float> vector3210(std::vector<float> data, int A, int B, int C, int D){
  assert(data.size()==A*B*C*D);
  std::vector<float> temp_data = data;
  for(int a=0;a<A;a++)
  for(int b=0;b<B;b++)
  for(int c=0;c<C;c++)
  for(int d=0;d<D;d++){
    temp_data.at(d*(A*B*C)+(c*B+b)*A+a) = data.at(a*(B*C*D)+(b*C+c)*D+d);
  }
  return temp_data;
}

